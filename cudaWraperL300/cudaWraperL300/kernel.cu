#include "hip/hip_runtime.h"
#pragma once

#include "cudaWrapperL300.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <vector>
#include <algorithm>

#define MINIMUN_VAL(x_,y_) (x_>y_)?x_:y_

#ifndef __HIPCC__
struct dim3 {
    dim3(int x_, int y_, int z_) :x(x_), y(y_), z(z_) {}
    int x;
    int y;
    int z;
};
#endif

#ifdef __HIPCC__

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    c[i] = a[i] + b[i];
}

__global__ void rescale_kernel(const unsigned short* input,
    float* output,
    const float* fractionalSamples,
    const float* wholeSamples,
    const unsigned int inputLength,
    const unsigned int outputLength)
{
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    float interpSample;
    int write_offset = j * outputLength;
    int read_offset = j * inputLength;
    int input_pos;
    unsigned int sampleIndex;
    sampleIndex = (int)wholeSamples[i];

    if ((sampleIndex + 1) > inputLength) {
        return;
    }

    input_pos = read_offset + sampleIndex;
    interpSample = (float)(input[input_pos + 1] - input[input_pos]) *
        fractionalSamples[i];
    interpSample = interpSample + input[input_pos];

    // Apply the passed in window function and set the output
    output[i + write_offset] = interpSample;
}

#endif

unsigned short* rescaleInputMemoryObject(0);
float* rescaleOutputMemObj(0);
float* fracSamplesMemObj(0);
float* wholeSamplesMemObj(0);

// Helper function for using CUDA to add vectors in parallel.
bool addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

 //    dim3 blocksPerGrid(size / 256, 1, 1);
    dim3 blocksPerGrid(MINIMUN_VAL(size/256,1), 1, 1);
    dim3 threadsPerBlock(256, 1, 1);

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
#ifdef __HIPCC__
    addKernel<<<blocksPerGrid, threadsPerBlock >>>(dev_c, dev_a, dev_b);
#endif

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus == hipSuccess;
}

bool resetCuda() {
    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    hipError_t cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }
    return (hipSuccess == cudaStatus);
}

bool addTwoVectors(int* c, const int* a, const int* b, unsigned int size)
{
    bool success{ false };

    success = addWithCuda(c, a, b, size);

    return success;
}
bool allocateInputMemObj(unsigned short* devMem, unsigned int linesPerFrame, unsigned int recordLength)
{
    hipError_t cudaStatus;

    const int size = linesPerFrame * recordLength * sizeof(unsigned short);
    cudaStatus = hipMalloc((void**)&devMem, size);
    return cudaStatus == hipSuccess;
}

bool initializeInputMemObj(unsigned short* devMem, unsigned short* data, unsigned int size)
{
    hipError_t cudaStatus;
    cudaStatus = hipMemcpy(devMem, data, size, hipMemcpyHostToDevice);
    return cudaStatus != hipSuccess;
}

bool allocateOutputMemObj(float* devMem, unsigned int linesPerFrame, unsigned int rescalingDataLength)
{
    hipError_t cudaStatus;

    const int size = linesPerFrame * rescalingDataLength * sizeof(float);
    cudaStatus = hipMalloc((void**)&devMem, size);
    return cudaStatus == hipSuccess;
}

bool allocateFracSamplesMemObj(float* devMem, unsigned int rescalingDataLength)
{
    hipError_t cudaStatus;

    const int size = rescalingDataLength * sizeof(float);
    cudaStatus = hipMalloc((void**)&devMem, size);
    return cudaStatus == hipSuccess;
}

bool initializeFracSamplesMemObj(float* devMem, float* data, unsigned int size)
{
    if (data) {
        hipError_t cudaStatus;
        cudaStatus = hipMemcpy(devMem, data, size, hipMemcpyHostToDevice);
        return cudaStatus != hipSuccess;
    }
    return false;
}

bool allocateWholeSamplesMemObj(float* devMem, unsigned int rescalingDataLength)
{
    hipError_t cudaStatus;

    const int size = rescalingDataLength * sizeof(float);
    cudaStatus = hipMalloc((void**)&devMem, size);
    return cudaStatus == hipSuccess;
}

bool initializeWholeSamplesMemObj(float* devMem, float* data, unsigned int size)
{
    if (data) {
        hipError_t cudaStatus;
        cudaStatus = hipMemcpy(devMem, data, size, hipMemcpyHostToDevice);
        return cudaStatus != hipSuccess;
    }
    return false;
}


bool cudaRescale(float* output, unsigned short* data, unsigned int inputSize,
    float* wholeSamples,
    float* fractionalSamples,
    char* errorMsg,
    unsigned int linesPerFrame, unsigned int recordLength, unsigned int rescalingDataLength)
{
    bool success{ false };

    if (data && inputSize && wholeSamples && fractionalSamples) {
        success = true;
        if (errorMsg) {
            errorMsg[0] = 0;
        }
    }

    if (success) {
        success = allocateInputMemObj(rescaleInputMemoryObject, linesPerFrame, recordLength);
    }
    else {
        if (errorMsg) {
            sprintf(errorMsg, "Invalid arguments");
        }
        return false;
    }

    if (success) {
        success = initializeInputMemObj(rescaleInputMemoryObject, data, inputSize);
    } 
    else {
        if (errorMsg) {
            sprintf(errorMsg, "failed to allocate rescaleInputMemoryObject");
        }
        return false;
    }
    if (success) {
        success = allocateOutputMemObj(rescaleOutputMemObj, linesPerFrame, rescalingDataLength);
    }
    else {
        if (errorMsg) {
            sprintf(errorMsg, "failed to initialize rescaleInputMemoryObject");
        }
        return false;
    }

    if (success) {
        success = allocateFracSamplesMemObj(fracSamplesMemObj, rescalingDataLength);
    }
    else {
        if (errorMsg) {
            sprintf(errorMsg, "failed to allocate rescaleOutputMemObj");
        }
        return false;
    }

    if (success) {
        success = initializeFracSamplesMemObj(fracSamplesMemObj, fractionalSamples, rescalingDataLength);
    }
    else {
        if (errorMsg) {
            sprintf(errorMsg, "failed to allocate fracSamplesMemObj");
        }
        return false;
    }

    if (success) {
        success = allocateWholeSamplesMemObj(wholeSamplesMemObj, rescalingDataLength);
    }
    else {
        if (errorMsg) {
            sprintf(errorMsg, "failed to initialize fracSamplesMemObj");
        }
        return false;
    }

    if (success) {
        success = initializeWholeSamplesMemObj(wholeSamplesMemObj, wholeSamples, rescalingDataLength);
    }
    else {
        if (errorMsg) {
            sprintf(errorMsg, "failed to allocate wholeSamplesMemObj");
        }
        return false;
    }

    if (!success) {
        if (errorMsg) {
            sprintf(errorMsg, "failed to initialize wholeSamplesMemObj");
        }
    }


    dim3 blocksPerGrid(inputSize/16, inputSize/16, 1);
    dim3 threadsPerBlock(16, 16, 1);

#ifdef __HIPCC__
    rescale_kernel << <blocksPerGrid, threadsPerBlock >> > (
        rescaleInputMemoryObject, rescaleOutputMemObj, 
        fracSamplesMemObj, wholeSamplesMemObj,
        recordLength, rescalingDataLength
        );
#endif
    // Check for any errors launching the kernel
    hipError_t cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
    }

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
    }

    // Copy output vector from GPU buffer to host memory.
    if (output) {
        cudaStatus = hipMemcpy(output, rescaleOutputMemObj, inputSize * sizeof(float), hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
        }
    }

//    success = cudaStatus == hipSuccess;

    if (success) {
        hipFree(rescaleInputMemoryObject);
    }

    return success;
}

