#include "hip/hip_runtime.h"
#pragma once

#include "cudaWrapperL300.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <vector>
#include <algorithm>

#define MINIMUN_VAL(x_,y_) (x_>y_)?x_:y_

#ifndef __HIPCC__
struct dim3 {
    dim3(int x_, int y_, int z_) :x(x_), y(y_), z(z_) {}
    int x;
    int y;
    int z;
};
#endif

#ifdef __HIPCC__

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    c[i] = a[i] + b[i];
}

__global__ void matrixAddKernel(int c[][4], const int a[][4], const int b[][4])
{
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    c[i][j] = a[i][j] + b[i][j];
}

__global__ void matrixAddKernel(int c[], const int a[], const int b[])
{
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    c[4*j + i] = a[4 * j + i] + b[4 * j + i];
}

#endif

bool matrix4by4Add(int c[][4], const int a[][4], const int b[][4], const unsigned int size)
{
    int* dev_a = 0;
    int* dev_b = 0;;
    int* dev_c = 0;

    int va[16];
    int vb[16];
    int vc[16];

    for (int i = 0; i < 4; ++i) {
        for (int j = 0; j < 4; ++j) {
            va[i + 4 * j] = a[i][j];
            vb[i + 4 * j] = b[i][j];
        }
    }

    hipError_t cudaStatus;

    dim3 blocksPerGrid(1, 1, 1);
    dim3 threadsPerBlock(4, 4, 1);
 //   unsigned int size = N * N * sizeof(int);

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, va, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "dev_a hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, vb, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "dev_b hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
#ifdef __HIPCC__
    matrixAddKernel << <blocksPerGrid, threadsPerBlock >> > (dev_c, dev_a, dev_b);
#endif

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(vc, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "dev_c hipMemcpy failed!");
        goto Error;
    }

    for (int i = 0; i < 4; ++i) {
        for (int j = 0; j < 4; ++j) {
            c[i][j] = vc[i + 4 * j];
        }
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);

    return cudaStatus == hipSuccess;
}


// Helper function for using CUDA to add vectors in parallel.
bool addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

 //    dim3 blocksPerGrid(size / 256, 1, 1);
    dim3 blocksPerGrid(MINIMUN_VAL(size/256,1), 1, 1);
    dim3 threadsPerBlock(256, 1, 1);

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
#ifdef __HIPCC__
    addKernel<<<blocksPerGrid, threadsPerBlock >>>(dev_c, dev_a, dev_b);
#endif

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus == hipSuccess;
}

bool resetCuda() {
    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    hipError_t cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }
    return (hipSuccess == cudaStatus);
}

bool addTwoVectors(int* c, const int* a, const int* b, unsigned int size)
{
    bool success{ false };

    success = addWithCuda(c, a, b, size);

    return success;
}
