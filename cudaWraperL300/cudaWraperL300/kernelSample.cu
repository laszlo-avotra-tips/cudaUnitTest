#include "hip/hip_runtime.h"
#pragma once

#include "cudaWrapperL300.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <vector>
#include <algorithm>

#define MINIMUN_VAL(x_,y_) (x_>y_)?x_:y_

#ifndef __HIPCC__
struct dim3 {
    dim3(int x_, int y_, int z_) :x(x_), y(y_), z(z_) {}
    int x;
    int y;
    int z;
};
#endif

#ifdef __HIPCC__

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    c[i] = a[i] + b[i];
}

__global__ void matrixAdd(float c[6][6], float a[6][6], float b[6][6])
{
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    c[i][j] = a[i][j] + b[i][j];
}

#endif

bool matrixAdd(float* c, const float* a, const float* b, const size_t N = 6)
{
    float* dev_a = 0;
    float* dev_b = 0;
    float* dev_c = 0;
    hipError_t cudaStatus;

    //    dim3 blocksPerGrid(size / 256, 1, 1);
    dim3 blocksPerGrid(MINIMUN_VAL(N / 4, 1), MINIMUN_VAL(N / 4, 1), 1);
    dim3 threadsPerBlock(4, 4, 1);
    size_t size = N * N * sizeof(float);

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
#ifdef __HIPCC__
    matrixAdd << <blocksPerGrid, threadsPerBlock >> > (dev_c, dev_a, dev_b);
#endif

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);

    return cudaStatus == hipSuccess;
}


// Helper function for using CUDA to add vectors in parallel.
bool addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

 //    dim3 blocksPerGrid(size / 256, 1, 1);
    dim3 blocksPerGrid(MINIMUN_VAL(size/256,1), 1, 1);
    dim3 threadsPerBlock(256, 1, 1);

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
#ifdef __HIPCC__
    addKernel<<<blocksPerGrid, threadsPerBlock >>>(dev_c, dev_a, dev_b);
#endif

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus == hipSuccess;
}

bool resetCuda() {
    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    hipError_t cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }
    return (hipSuccess == cudaStatus);
}

bool addTwoVectors(int* c, const int* a, const int* b, unsigned int size)
{
    bool success{ false };

    success = addWithCuda(c, a, b, size);

    return success;
}
