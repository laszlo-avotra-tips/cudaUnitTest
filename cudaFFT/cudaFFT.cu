#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2017 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/* Example showing the use of CUFFT for fast 1D-convolution using FFT. */

// includes, system
#include <algorithm>
#include <iostream>
#include <complex>
#include <vector>

// includes, project
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime_api.h>

#ifndef __HIPCC__
struct dim3 {
    dim3(int x_, int y_, int z_) :x(x_), y(y_), z(z_) {}
    int x;
    int y;
    int z;
};
#endif

// Complex data type
using Complex = std::complex<float>;
using ComplexVector = std::vector<Complex>;

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest(int argc, char **argv);
void addjustCoefficientMagnitude(Complex* h_data, size_t dataSize);
int isOriginalEqualToTheTransformedAndInverseTransformenData(
    const Complex* original, const Complex* transformed, size_t dataSize);
void printTheData(const Complex* original, const Complex* transformed, size_t dataSize);

//// The filter size is assumed to be a number smaller than the signal size

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv) { 
    runTest(argc, argv); 
}

////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
void runTest(int argc, char **argv) {
  std::cout << "[cudaFFT] is starting..." << std::endl;

  findCudaDevice(argc, (const char **)argv);

  const size_t SIGNAL_SIZE{ 256 };

  // Allocate host memory for the signal
  Complex* h_signal = new Complex[SIGNAL_SIZE];
  Complex* h_signal_fft_ifft = new Complex[SIGNAL_SIZE];

  // Initialize the memory for the signal
  for (unsigned int i = 0; i < SIGNAL_SIZE; ++i) {
      h_signal[i] = { rand() / static_cast<float>(RAND_MAX), 0 };
      h_signal_fft_ifft[i] = { float(i), 1000.f * i };
  }

  int mem_size = sizeof(Complex) * SIGNAL_SIZE;

  // Allocate device memory for signal
  Complex *d_signal;
  checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_signal), mem_size));

  // Copy host memory to device
  checkCudaErrors(hipMemcpy(d_signal, h_signal, mem_size, hipMemcpyHostToDevice));

  // CUFFT plan simple API
  hipfftHandle plan;
  checkCudaErrors(hipfftPlan1d(&plan, mem_size, HIPFFT_C2C, 1));

  // Transform signal and kernel
  std::cout << "Transforming signal cufftExecC2" << std::endl;
  checkCudaErrors(hipfftExecC2C(plan, reinterpret_cast<hipfftComplex *>(d_signal),
                               reinterpret_cast<hipfftComplex *>(d_signal),
                               HIPFFT_FORWARD));
  //h_signal has the original coefficients
  //d_signal has the direct FFT coefficients

  // Check if kernel execution generated and error
  getLastCudaError("Kernel execution failed [ ComplexPointwiseMulAndScale ]");
  // Transform signal back
  std::cout << "Transforming signal back hipfftExecC2C" << std::endl;

  checkCudaErrors(hipfftExecC2C(plan, reinterpret_cast<hipfftComplex *>(d_signal),
                               reinterpret_cast<hipfftComplex *>(d_signal),
                               HIPFFT_BACKWARD));
  //h_signal has the original coefficients
  //d_signal has the FFT --> iFFT coefficients

  // Copy device memory to host
  checkCudaErrors(hipMemcpy(h_signal_fft_ifft, d_signal, mem_size,
                             hipMemcpyDeviceToHost));
  //h_signal has the original coefficients
  //h_signal_fft_ifft has the FFT --> iFFT coefficients

  // check result
  int iTestResult = 0;

  //result scaling
  addjustCoefficientMagnitude(h_signal_fft_ifft, SIGNAL_SIZE);

  iTestResult = isOriginalEqualToTheTransformedAndInverseTransformenData(h_signal, h_signal_fft_ifft, SIGNAL_SIZE);

  printTheData(h_signal, h_signal_fft_ifft, 10);

  // Destroy CUFFT context
  checkCudaErrors(hipfftDestroy(plan));

  // cleanup memory
  delete h_signal;
  delete h_signal_fft_ifft;

  checkCudaErrors(hipFree(d_signal));

  exit((iTestResult == 0) ? EXIT_SUCCESS : EXIT_FAILURE);
}

void addjustCoefficientMagnitude(Complex* h_data, size_t dataSize)
{
    for (size_t i = 0; i < dataSize; ++i) {
        h_data[i] = { h_data[i].real() / 8.0f / dataSize, 0 };
    }
}

int isOriginalEqualToTheTransformedAndInverseTransformenData(
    const Complex* original, const Complex* transformed, size_t dataSize)
{
    int iTestResult = 0;
    for (int i = 0; i < dataSize; ++i) {
        if (std::abs(transformed[i].real() - original[i].real()) > abs(original[i].real() * 1e-5f))
            iTestResult += 1;
    }
    return iTestResult;
}

void printTheData(const Complex* original, const Complex* transformed, size_t dataSize)
{
    std::cout << "The first " << dataSize << " real values:" << std::endl;
    for (int i = 0; i < dataSize; ++i) {
        std::cout << original[i].real() << " ";
    }
    std::cout << std::endl;
    for (int i = 0; i < dataSize; ++i) {
        std::cout << transformed[i].real() << " ";
    }
    std::cout << std::endl;
}
