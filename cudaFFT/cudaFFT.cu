#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2017 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/* Example showing the use of CUFFT for fast 1D-convolution using FFT. */

// includes, system
#include <algorithm>
#include <iostream>
#include <memory>

#include <cudaFFTwrapper.h>

#ifndef __HIPCC__
struct dim3 {
    dim3(int x_, int y_, int z_) :x(x_), y(y_), z(z_) {}
    int x;
    int y;
    int z;
};
#endif

// Complex data type
using Complex = std::complex<float>;

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest(int argc, char **argv);
void addjustCoefficientMagnitude(Complex* h_data, size_t dataSize) noexcept;
int isOriginalEqualToTheTransformedAndInverseTransformenData(
    const Complex* original, const Complex* transformed, size_t dataSize) noexcept;
void printTheData(const Complex* original, const Complex* transformed, size_t dataSize);
void initializeTheSignals(Complex* fft, Complex* invfft, size_t dataSize) noexcept;

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv) { 
    runTest(argc, argv); 
}

////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
void runTest(int argc, char **argv) {
  std::cout << "[cudaFFT] is starting..." << std::endl;

  /*findCudaDevice(argc, (const char **)argv);*/

  const size_t SIGNAL_SIZE{ 256 };

  // Allocate host memory for the signal
  //Complex* h_signal = new Complex[SIGNAL_SIZE];
  std::unique_ptr<Complex[]> h_signal = std::make_unique<Complex[]>(SIGNAL_SIZE);

  //Complex* h_signal_fft_ifft = new Complex[SIGNAL_SIZE];
  std::unique_ptr<Complex[]> h_signal_fft_ifft = std::make_unique<Complex[]>(SIGNAL_SIZE);
  initializeTheSignals(h_signal.get(), h_signal_fft_ifft.get(), SIGNAL_SIZE);
  
  ComputeTheFFT(h_signal.get(), h_signal_fft_ifft.get(), SIGNAL_SIZE);

  // check result
  int iTestResult = 0;

  //result scaling
  addjustCoefficientMagnitude(h_signal_fft_ifft.get(), SIGNAL_SIZE);

  iTestResult = isOriginalEqualToTheTransformedAndInverseTransformenData(h_signal.get(), h_signal_fft_ifft.get(), SIGNAL_SIZE);

  printTheData(h_signal.get(), h_signal_fft_ifft.get(), 10);


  // cleanup memory
  //delete h_signal;
  //delete h_signal_fft_ifft;

  exit((iTestResult == 0) ? EXIT_SUCCESS : EXIT_FAILURE);
}

void addjustCoefficientMagnitude(Complex* h_data, size_t dataSize) noexcept
{
    if (h_data) {
        for (size_t i = 0; i < dataSize; ++i) {
            h_data[i] = { h_data[i].real() / 8.0f / dataSize, 0 };
        }
    }
}

int isOriginalEqualToTheTransformedAndInverseTransformenData(
    const Complex* original, const Complex* transformed, size_t dataSize) noexcept
{
    int iTestResult = 1;
    if (original && transformed) {
        iTestResult = 0;
        for (int i = 0; i < dataSize; ++i) {
            if (std::abs(transformed[i].real() - original[i].real()) > abs(original[i].real() * 1e-5f))
                iTestResult += 1;
        }
    }
    return iTestResult;
}

void printTheData(const Complex* original, const Complex* transformed, size_t dataSize)
{
    std::cout << "The first " << dataSize << " real values:" << std::endl;
    if (original) {
        for (int i = 0; i < dataSize; ++i) {
            std::cout << original[i].real() << " ";
        }
        std::cout << std::endl;
    }
    if (transformed) {
        for (int i = 0; i < dataSize; ++i) {
            std::cout << transformed[i].real() << " ";
        }
        std::cout << std::endl;
    }
}

void initializeTheSignals(Complex* fft, Complex* invfft, size_t dataSize) noexcept
{
    for (size_t i = 0; i < dataSize; ++i) {
        if(fft)
            fft[i] = { rand() / static_cast<float>(RAND_MAX), 0 };
        if(invfft)
            invfft[i] = { float(i), 1000.f * i };
    }
}


