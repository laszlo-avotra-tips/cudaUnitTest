#include "hip/hip_runtime.h"
#pragma once

#include "cudaWrapperL300.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <vector>
#include <algorithm>

#define MINIMUN_VAL(x_,y_) (x_>y_)?x_:y_

#ifndef __HIPCC__
struct dim3 {
    dim3(int x_, int y_, int z_) :x(x_), y(y_), z(z_) {}
    int x;
    int y;
    int z;
};
#endif

#ifdef __HIPCC__

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    c[i] = a[i] + b[i];
}
#endif

// Helper function for using CUDA to add vectors in parallel.
bool addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

 //    dim3 blocksPerGrid(size / 256, 1, 1);
    dim3 blocksPerGrid(MINIMUN_VAL(size/256,1), 1, 1);
    dim3 threadsPerBlock(256, 1, 1);

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
#ifdef __HIPCC__
    addKernel<<<blocksPerGrid, threadsPerBlock >>>(dev_c, dev_a, dev_b);
#endif

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus == hipSuccess;
}

bool resetCuda() {
    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    hipError_t cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }
    return (hipSuccess == cudaStatus);
}

bool addTwoVectors(int* c, const int* a, const int* b, unsigned int size)
{
    bool success{ false };

    success = addWithCuda(c, a, b, size);

    return success;
}
bool allocateInputMemObj(unsigned short* devMem, unsigned int linesPerFrame, unsigned int recordLength)
{
    hipError_t cudaStatus;

    const int size = linesPerFrame * recordLength * sizeof(unsigned short);
    cudaStatus = hipMalloc((void**)&devMem, size);
    return cudaStatus == hipSuccess;
}

bool initializeInputMemObj(unsigned short* devMem, unsigned short* data, unsigned int size)
{
    hipError_t cudaStatus;
    cudaStatus = hipMemcpy(devMem, data, size, hipMemcpyHostToDevice);
    return cudaStatus != hipSuccess;
}

bool allocateOutputMemObj(float* devMem, unsigned int linesPerFrame, unsigned int rescalingDataLength)
{
    hipError_t cudaStatus;

    const int size = linesPerFrame * rescalingDataLength * sizeof(float);
    cudaStatus = hipMalloc((void**)&devMem, size);
    return cudaStatus == hipSuccess;
}

bool allocateFracSamplesMemObj(float* devMem, unsigned int rescalingDataLength)
{
    hipError_t cudaStatus;

    const int size = rescalingDataLength * sizeof(float);
    cudaStatus = hipMalloc((void**)&devMem, size);
    return cudaStatus == hipSuccess;
}

bool initializeFracSamplesMemObj(float* devMem, float* data, unsigned int size)
{
    if (data) {
        hipError_t cudaStatus;
        cudaStatus = hipMemcpy(devMem, data, size, hipMemcpyHostToDevice);
        return cudaStatus != hipSuccess;
    }
    return false;
}

bool cudaRescale(unsigned short* data, unsigned int size,
    float* wholeSamples,
    float* fractionalSamples,
    char* errorMsg,
    unsigned int linesPerFrame, unsigned int recordLength, unsigned int rescalingDataLength)
{
    bool success{ false };

    if (data && size) {
        success = true;
        if (errorMsg) {
            *errorMsg = 0;
        }
    } else {
        if (errorMsg) {
            sprintf(errorMsg, "Invalid arguments");
        }
    }

    unsigned short* rescaleInputMemoryObject(0);
    float* rescaleOutputMemObj(0);
    float* fracSamplesMemObj(0);

    if (success) {
        success = allocateInputMemObj(rescaleInputMemoryObject, linesPerFrame, recordLength);
    }

    if (success) {
        success = initializeInputMemObj(rescaleInputMemoryObject, data, size);
    } 

    if (success) {
        hipFree(rescaleInputMemoryObject);
    }

    if (success) {
        success = allocateOutputMemObj(rescaleOutputMemObj, linesPerFrame, rescalingDataLength);
    }

    if (success) {
        success = allocateFracSamplesMemObj(fracSamplesMemObj, rescalingDataLength);
    }

    if (success) {
        success = initializeFracSamplesMemObj(fracSamplesMemObj, fractionalSamples, rescalingDataLength);
    }

    return success;
}

